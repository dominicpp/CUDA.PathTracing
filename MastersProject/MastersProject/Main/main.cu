#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>

#include <fstream>
#include <iostream>
#include <chrono>

#include "../Utils/vec3.cuh"
#include "../Ray/ray.cuh"
#include "../Hit/group.cuh"
#include "../Camera/camera.cuh"
#include "../Material/material.cuh"
#include "../Material/diffuseMaterial.cuh"
#include "../Material/mirrorMaterial.cuh"
#include "../Material/polishedMetalMaterial.cuh"
#include "../Objects/sphere.cuh"


__device__ Vec3 calculateRadiance(const Ray& ray, Shape** scene, int depth, hiprandStateXORWOW* state)
{
    Ray tempRay = ray;
    Vec3 attenuation = Vec3(1.0f, 1.0f, 1.0f);
    int bounces = 0;

    while (bounces < depth)
    {
        RecordHit hit;
        if ((*scene)->hitIntersect(tempRay, 0.001f, FLT_MAX, hit))
        {
            Ray scattered;
            Vec3 albedo = hit.material->albedo();
            if (hit.material->scatteredRay(tempRay, hit, scattered, state))
            {
                attenuation = attenuation * albedo;
                tempRay = scattered;
                bounces++;
                continue;
            }
            else return Vec3(0.0f, 0.0f, 0.0f);
        }
        // background
        return attenuation;
    }
    return Vec3(0.0f, 0.0f, 0.0f);
}

//#############################################
__global__ void render_init(int width, int height, hiprandStateXORWOW* state)
{
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int offsetx = blockIdx.x * blockDim.x;
    int offsety = blockIdx.y * blockDim.y;
    int gidx = tidx + offsetx;
    int gidy = tidy + offsety;
    if ((gidx >= width) || (gidy >= height)) return;
    int pixelIndex = gidy * width + gidx;
    hiprand_init(2023, pixelIndex, 0, &state[pixelIndex]);
}
//#############################################

__global__ void raytrace(Vec3* buffer, int width, int height, Camera** camera, Shape** scene, hiprandStateXORWOW* state, int sample, float gamma)
{
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int offsetx = blockIdx.x * blockDim.x;
    int offsety = blockIdx.y * blockDim.y;
    int gidx = tidx + offsetx;
    int gidy = tidy + offsety;
    if ((gidx >= width) || (gidy >= height)) return;
    int pixelIndex = gidy * width + gidx;

    hiprandStateXORWOW tempState = state[pixelIndex];
    Vec3 color(0, 0, 0);
    for (int x = 0; x < sample; ++x)
    {
        for (int y = 0; y < sample; ++y)
        {
            float rx = hiprand_uniform(&tempState);
            float ry = hiprand_uniform(&tempState);
            float sx = (gidx + (x + rx) / sample) / float(width);
            float sy = (gidy + (y + ry) / sample) / float(height);
            Ray ray = (*camera)->generateRay(sx, sy);
            color = color + calculateRadiance(ray, scene, 15, &tempState);
        }
    }

    Vec3 setPixel = color;
    setPixel = color / float(sample * sample);
    setPixel[0] = pow(setPixel[0], 1 / gamma);
    setPixel[1] = pow(setPixel[1], 1 / gamma);
    setPixel[2] = pow(setPixel[2], 1 / gamma);
    buffer[pixelIndex] = setPixel;
    state[pixelIndex] = tempState;
}

__global__ void create_world(Shape** d_list, Shape** d_world, Camera** d_camera)
{
    d_list[0] = new Sphere(Vec3(0.0, 0.0, -1.0), 0.5, new Diffuse(Vec3(0.2, 0.6, 0.8))); // center diffuse sphere
    d_list[1] = new Sphere(Vec3(0.0, 0.0, 1.5), 0.5, new Diffuse(Vec3(1.0, 0.0, 1.0))); // behind camera diffuse sphere
    d_list[2] = new Sphere(Vec3(-0.20, -0.45, -0.65), 0.05, new Diffuse(Vec3(1.0, 0.45, 0.5))); // pink diffuse sphere infront of center sphere
    d_list[3] = new Sphere(Vec3(0.78, -0.15, -1.0), 0.3, new PolishedMetal(Vec3(1.0, 1.0, 1.0), 0.23)); // polished metal sphere right from center sphere
    d_list[4] = new Sphere(Vec3(-0.78, -0.15, -1.0), 0.3, new Diffuse(Vec3(1.0, 0.0, 0.0))); // red diffuse sphere
    d_list[5] = new Sphere(Vec3(0.75, -0.23, -0.48), 0.1, new Mirror(Vec3(1.0, 1.0, 1.0))); // mirror sphere down right
    d_list[6] = new Sphere(Vec3(-0.75, -0.23, -0.48), 0.1, new Mirror(Vec3(1.0, 1.0, 1.0))); // mirror sphere down left
    d_list[7] = new Sphere(Vec3(0.29, 0.2, -0.39), 0.05, new Diffuse(Vec3(0.2, 0.8, 0.2))); // green sphere up right
    d_list[8] = new Sphere(Vec3(-0.29, 0.2, -0.39), 0.05, new PolishedMetal(Vec3(1.0, 1.0, 1.0), 1.0)); // polished metal sphere up left
    d_list[9] = new Sphere(Vec3(0.0, -100.5, -1.0), 100, new Diffuse(Vec3(0.85, 0.85, 0.85))); // plane sphere
    d_list[10] = new Sphere(Vec3(-0.43, -0.40, -0.85), 0.05, new Mirror(Vec3(1.0, 0.0, 1.0))); // tiny purple mirror sphere 
    d_list[11] = new Sphere(Vec3(0.40, -0.40, -0.75), 0.09, new Mirror(Vec3(1.0, 1.0, 0.0))); // yellow mirror sphere
    d_list[12] = new Sphere(Vec3(-0.15, 0.21, -0.56), 0.06, new Diffuse(Vec3(0.2, 0.8, 0.6))); // aqua sphere on blue sphere

    *d_world = new Group(d_list, 13);
    *d_camera = new Camera(4.0f, 2.0f);
}

int main()
{
    // resolution in x & y dimension / number of threads for each dimension
    int nx = 1200;
    int ny = 600;
    // number of (thread-)blocks in x & y dimension
    int tx = 32;
    int ty = 32;
    int sample = 10; // rays per pixel -> in fact 32x32 with Stratified Sampling
    float gamma = 2.2f; // corrected gamma value

    int allPixels = nx * ny;
    float bufferSize = allPixels * sizeof(Vec3);

    std::ofstream out("doc/test_albedo2.ppm");
    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << sample * sample << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    // ########## CUDA MEMORY ALLOCATION
    hiprandStateXORWOW* d_state; // Random Number Generator
    hipMallocManaged((void**)&d_state, allPixels * sizeof(hiprandStateXORWOW));
    Vec3* d_buffer;
    hipMallocManaged((void**)&d_buffer, bufferSize);
    Shape** d_objects;
    hipMallocManaged((void**)&d_objects, 12 * sizeof(Shape*));
    Shape** d_scene;
    hipMallocManaged((void**)&d_scene, sizeof(Shape*));
    Camera** d_camera;
    hipMallocManaged((void**)&d_camera, sizeof(Camera*));
    // ##########

    dim3 grid(nx / tx + 1, ny / ty + 1, 1);
    dim3 block(tx, ty, 1);

    auto a = std::chrono::high_resolution_clock::now();
    // KERNEL 1
    create_world << <1, 1 >> > (d_objects, d_scene, d_camera);
    hipGetLastError();
    hipDeviceSynchronize();
    //#############################################
    // KERNEL 2
    render_init << <grid, block >> > (nx, ny, d_state);
    hipGetLastError();
    hipDeviceSynchronize();
    //#############################################
    // KERNEL 3
    raytrace << <grid, block >> > (d_buffer, nx, ny, d_camera, d_scene, d_state, sample, gamma);
    hipGetLastError();
    hipDeviceSynchronize();
    auto b = std::chrono::high_resolution_clock::now();
    std::cerr << "\nRendering took: " << std::chrono::duration_cast<std::chrono::seconds>(b - a).count() << " seconds\n";

    out << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--)
    {
        for (int i = 0; i < nx; i++)
        {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * d_buffer[pixel_index][0]);
            int ig = int(255.99 * d_buffer[pixel_index][1]);
            int ib = int(255.99 * d_buffer[pixel_index][2]);
            out << ir << " " << ig << " " << ib << "\n";
        }
    }

    // free memory on device
    hipFree(d_camera);
    hipFree(d_scene);
    hipFree(d_objects);
    hipFree(d_state);
    hipFree(d_buffer);
    // remove all device allocations
    hipDeviceReset();
}